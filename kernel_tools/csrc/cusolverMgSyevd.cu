/*
 * Copyright 2020 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO LICENSEE:
 *
 * This source code and/or documentation ("Licensed Deliverables") are
 * subject to NVIDIA intellectual property rights under U.S. and
 * international Copyright laws.
 *
 * These Licensed Deliverables contained herein is PROPRIETARY and
 * CONFIDENTIAL to NVIDIA and is being provided under the terms and
 * conditions of a form of NVIDIA software license agreement by and
 * between NVIDIA and Licensee ("License Agreement") or electronically
 * accepted by Licensee.  Notwithstanding any terms or conditions to
 * the contrary in the License Agreement, reproduction or disclosure
 * of the Licensed Deliverables to any third party without the express
 * written consent of NVIDIA is prohibited.
 *
 * NOTWITHSTANDING ANY TERMS OR CONDITIONS TO THE CONTRARY IN THE
 * LICENSE AGREEMENT, NVIDIA MAKES NO REPRESENTATION ABOUT THE
 * SUITABILITY OF THESE LICENSED DELIVERABLES FOR ANY PURPOSE.  IT IS
 * PROVIDED "AS IS" WITHOUT EXPRESS OR IMPLIED WARRANTY OF ANY KIND.
 * NVIDIA DISCLAIMS ALL WARRANTIES WITH REGARD TO THESE LICENSED
 * DELIVERABLES, INCLUDING ALL IMPLIED WARRANTIES OF MERCHANTABILITY,
 * NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.
 * NOTWITHSTANDING ANY TERMS OR CONDITIONS TO THE CONTRARY IN THE
 * LICENSE AGREEMENT, IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY
 * SPECIAL, INDIRECT, INCIDENTAL, OR CONSEQUENTIAL DAMAGES, OR ANY
 * DAMAGES WHATSOEVER RESULTING FROM LOSS OF USE, DATA OR PROFITS,
 * WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE OR OTHER TORTIOUS
 * ACTION, ARISING OUT OF OR IN CONNECTION WITH THE USE OR PERFORMANCE
 * OF THESE LICENSED DELIVERABLES.
 *
 * U.S. Government End Users.  These Licensed Deliverables are a
 * "commercial item" as that term is defined at 48 C.F.R. 2.101 (OCT
 * 1995), consisting of "commercial computer software" and "commercial
 * computer software documentation" as such terms are used in 48
 * C.F.R. 12.212 (SEPT 1995) and is provided to the U.S. Government
 * only as a commercial end item.  Consistent with 48 C.F.R.12.212 and
 * 48 C.F.R. 227.7202-1 through 227.7202-4 (JUNE 1995), all
 * U.S. Government End Users acquire the Licensed Deliverables with
 * only those rights set forth herein.
 *
 * Any use of the Licensed Deliverables in individual and commercial
 * software must include, in the user documentation and internal
 * comments to the code, the above Disclaimer and U.S. Government End
 * Users Notice.
 */

#include <torch/extension.h>
#include <signal.h>

#include <cstdio>
#include <cstdlib>
#include <vector>

#include <hip/hip_runtime.h>
#include <hipsolver.h>

#include "cusolverMg_utils.h"
#include "cusolver_utils.h"

#include "kernel_tools.h"

// Class to manage the cuBLAS handle
class CuSolverMgHandle {
public:
    CuSolverMgHandle() {
        printf("create handle\n");
        hipsolverStatus_t status = cusolverMgCreate(&handle);
        if (status != HIPSOLVER_STATUS_SUCCESS) {
            throw std::runtime_error("Failed to create cuSolver handle");
        }
    }

    ~CuSolverMgHandle() {
        hipsolverStatus_t status = cusolverMgDestroy(handle);
        if (status != HIPSOLVER_STATUS_SUCCESS) {
            std::cerr << "Failed to destroy cuSolver handle" << std::endl;
        }
    }

    cusolverMgHandle_t get() const {
        return handle;
    }

private:
    cusolverMgHandle_t handle;
};

// Singleton pattern to ensure only one instance of CuSolverMgHandle
CuSolverMgHandle& get_cusolver_mg_handle() {
    static CuSolverMgHandle handle;
    return handle;
}

template <typename T>
void cusolverMgSyevd_workspace_template(
    int N,
    int num_devices,
    bool use_num_devices_visible,
    int64_t *workspace_elements,
    bool verbose
) {
    using data_type = T;

    cusolverMgHandle_t cusolverH = get_cusolver_mg_handle().get();

    const int MAX_NUM_DEVICES = 16;

    std::vector<int> deviceList(MAX_NUM_DEVICES);

    const int IA = 1;
    const int JA = 1;
    const int T_A = 256; /* tile size */
    const int lda = N;

    hipsolverEigMode_t jobz = HIPSOLVER_EIG_MODE_VECTOR;

    cudaLibMgMatrixDesc_t descrA = NULL;
    cudaLibMgGrid_t gridA = NULL;
    cusolverMgGridMapping_t mapping = CUDALIBMG_GRID_MAPPING_COL_MAJOR;

    int64_t lwork = 0; /* workspace: number of elements per device */

    int nbGpus = num_devices;
    if (use_num_devices_visible) {
        CUDA_CHECK( hipGetDeviceCount(&nbGpus) );

        nbGpus = (nbGpus < MAX_NUM_DEVICES) ? nbGpus : MAX_NUM_DEVICES;
        if (verbose) std::printf("\tThere are %d GPUs \n", nbGpus);
        for (int j = 0; j < nbGpus; j++) {
            deviceList[j] = j;
            hipDeviceProp_t prop;
            CUDA_CHECK( hipGetDeviceProperties(&prop, j) );
            if (verbose) std::printf("\tDevice %d, %s, cc %d.%d \n", j, prop.name, prop.major, prop.minor);
        }
    }

    CUSOLVER_CHECK( cusolverMgDeviceSelect(cusolverH, nbGpus, deviceList.data()) );
    CUSOLVER_CHECK( cusolverMgCreateDeviceGrid(&gridA, 1, nbGpus, deviceList.data(), mapping) );

    /* (global) A is N-by-N */
    CUSOLVER_CHECK(
        cusolverMgCreateMatrixDesc(
            &descrA, 
            N, /* nubmer of rows of (global) A */
            N,          /* number of columns of (global) A */
            N,          /* number or rows in a tile */
            T_A,        /* number of columns in a tile */
            traits<data_type>::cuda_data_type, 
            gridA
        )
    );

    CUSOLVER_CHECK( 
        cusolverMgSyevd_bufferSize(
            cusolverH, 
            jobz, 
            HIPBLAS_FILL_MODE_LOWER, /* only support lower mode */
            N, 
            NULL,
            IA,         /* base-1 */
            JA,         /* base-1 */
            descrA, 
            NULL,
            traits<data_type>::cuda_data_type,
            traits<data_type>::cuda_data_type, 
            &lwork
        )
    );

    if (verbose) std::printf("\tAllocate device workspace, lwork = %lld \n", static_cast<long long>(lwork));
    *workspace_elements = lwork;

    if (descrA != NULL) {
        CUSOLVER_CHECK( cusolverMgDestroyMatrixDesc(descrA) );
    }

    if (gridA != NULL) {
        CUSOLVER_CHECK( cusolverMgDestroyGrid(gridA) );
    }
}

template <typename T>
void cusolverMgSyevd_template(
    torch::Tensor a, 
    torch::Tensor d,
    bool verbose
) {

    size_t start_free_mem, end_free_mem, total_mem;
    CUDA_CHECK( hipMemGetInfo(&start_free_mem, &total_mem) );
    printf("start: %zu\n", start_free_mem);

    using data_type = T;

    cusolverMgHandle_t cusolverH = get_cusolver_mg_handle().get();

    /* maximum number of GPUs */
    const int MAX_NUM_DEVICES = 16;

    int nbGpus = 0;
    std::vector<int> deviceList(MAX_NUM_DEVICES);

    const int N = a.size(0);
    const int IA = 1;
    const int JA = 1;
    const int T_A = 256; /* tile size */
    const int lda = N;

    int info = 0;

    hipsolverEigMode_t jobz = HIPSOLVER_EIG_MODE_VECTOR;

    cudaLibMgMatrixDesc_t descrA = NULL;
    cudaLibMgGrid_t gridA = NULL;
    cusolverMgGridMapping_t mapping = CUDALIBMG_GRID_MAPPING_COL_MAJOR;

    int64_t lwork = 0; /* workspace: number of elements per device */

    if (verbose) std::printf("Step 1: Create Mg handle and select devices \n");
    CUDA_CHECK( hipGetDeviceCount(&nbGpus) );

    nbGpus = (nbGpus < MAX_NUM_DEVICES) ? nbGpus : MAX_NUM_DEVICES;
    if (verbose) std::printf("\tThere are %d GPUs \n", nbGpus);
    for (int j = 0; j < nbGpus; j++) {
        deviceList[j] = j;
        hipDeviceProp_t prop;
        CUDA_CHECK( hipGetDeviceProperties(&prop, j) );
        if (verbose) std::printf("\tDevice %d, %s, cc %d.%d \n", j, prop.name, prop.major, prop.minor);
    } 

    CUSOLVER_CHECK( cusolverMgDeviceSelect(cusolverH, nbGpus, deviceList.data()) );

    if (verbose) std::printf("step 2: Enable peer access.\n");
    CUDA_CHECK( enablePeerAccess(nbGpus, deviceList.data()) );

    if (verbose) std::printf("Step 5: Create matrix descriptors for A and D \n");
    CUSOLVER_CHECK( cusolverMgCreateDeviceGrid(&gridA, 1, nbGpus, deviceList.data(), mapping) );

    /* (global) A is N-by-N */
    CUSOLVER_CHECK(
        cusolverMgCreateMatrixDesc(
            &descrA, 
            N, /* nubmer of rows of (global) A */
            N,          /* number of columns of (global) A */
            N,          /* number or rows in a tile */
            T_A,        /* number of columns in a tile */
            traits<data_type>::cuda_data_type, gridA
        )
    );

    if (verbose) std::printf("Step 8: Allocate workspace space \n");
    CUSOLVER_CHECK( 
        cusolverMgSyevd_bufferSize(
            cusolverH, 
            jobz, 
            HIPBLAS_FILL_MODE_LOWER, /* only support lower mode */
            N, 
            NULL, //reinterpret_cast<void **>(array_d_A.data()), 
            IA,         /* base-1 */
            JA,         /* base-1 */
            descrA, 
            NULL, // reinterpret_cast<void *>(d_data), 
            traits<data_type>::cuda_data_type,
            traits<data_type>::cuda_data_type, 
            &lwork
        )
    );

    if (verbose) std::printf("\tAllocate device workspace, lwork = %lld \n", static_cast<long long>(lwork));

    if (verbose) std::printf("Step 6: Allocate distributed matrices A and D \n");
    std::vector<data_type *> array_d_A(nbGpus, nullptr);
    data_type *a_data = a.data_ptr<data_type>();
    data_type *d_data = d.data_ptr<data_type>();

    std::vector<data_type *> array_d_work(nbGpus, nullptr);

    hipError_t cuda_status = hipSuccess;
    hipsolverStatus_t cuda_solver_status = HIPSOLVER_STATUS_SUCCESS;

    // Hacky way to make sure on a bad status we still deallocate all the memory
    do {
        /* A := 0 */
        cuda_status = 
            createMat<data_type>(
                nbGpus, 
                deviceList.data(), 
                N, /* number of columns of global A */
                T_A,                          /* number of columns per column tile */
                lda,                          /* leading dimension of local A */
                array_d_A.data()
            );
        if (cuda_status != hipSuccess) break;

        if (verbose) std::printf("Step 7: Prepare data on devices \n");
        cuda_status = 
            memcpyH2D<data_type>(nbGpus, deviceList.data(), N, N,
                /* input */
                a_data, lda,
                /* output */
                N,                /* number of columns of global A */
                T_A,              /* number of columns per column tile */
                lda,              /* leading dimension of local A */
                array_d_A.data(), /* host pointer array of dimension nbGpus */
                IA, JA
            );
        if (cuda_status != hipSuccess) break;

        /* array_d_work[j] points to device workspace of device j */
        cuda_status = 
            workspaceAlloc(
                nbGpus, 
                deviceList.data(),
                sizeof(data_type) * lwork, /* number of bytes per device */
                reinterpret_cast<void **>(array_d_work.data())
            );
        if (cuda_status != hipSuccess) break;

        /* sync all devices */
        cuda_status = hipDeviceSynchronize();
        if (cuda_status != hipSuccess) break;

        if (verbose) std::printf("Step 9: Compute eigenvalues and eigenvectors \n");
        cuda_solver_status = 
            cusolverMgSyevd(
                cusolverH, 
                jobz, 
                HIPBLAS_FILL_MODE_LOWER, /* only support lower mode */
                N, 
                reinterpret_cast<void **>(array_d_A.data()),             /* exit: eigenvectors */
                IA, 
                JA, 
                descrA, 
                reinterpret_cast<void **>(d_data),        /* exit: eigenvalues */
                traits<data_type>::cuda_data_type, 
                traits<data_type>::cuda_data_type,
                reinterpret_cast<void **>(array_d_work.data()), 
                lwork, 
                &info /* host */
            );
        if (cuda_solver_status != HIPSOLVER_STATUS_SUCCESS) break;

        /* sync all devices */
        cuda_status = hipDeviceSynchronize();
        if (cuda_status != hipSuccess) break;

        /* check if SYEVD converges */
        if (0 > info) {
            // Break here and check info after everything is freed
            break;
        }

        if (verbose) std::printf("Step 10: Copy eigenvectors to A and eigenvalues to D\n");
        cuda_status = 
            memcpyD2H<data_type>(nbGpus, deviceList.data(), N, N,
                /* input */
                N,   /* number of columns of global A */
                T_A, /* number of columns per column tile */
                lda, /* leading dimension of local A */
                array_d_A.data(), IA, JA,
                /* output */
                a_data, /* N-y-N eigenvectors */
                lda
            );
        if (cuda_status != hipSuccess) break;
    } while(0);

    hipError_t destroyMat_status;
    hipError_t workspaceFree_status;

    if (verbose) std::printf("step 12: Free resources \n");
    destroyMat_status = 
        destroyMat(
            nbGpus, 
            deviceList.data(), 
            N,   /* number of columns of global A */
            T_A, /* number of columns per column tile */
            reinterpret_cast<void **>(array_d_A.data())
        );

    workspaceFree_status = workspaceFree(nbGpus, deviceList.data(), reinterpret_cast<void **>(array_d_work.data()));

    CUDA_CHECK(destroyMat_status);
    CUDA_CHECK(workspaceFree_status);

    if (descrA != NULL) {
        CUSOLVER_CHECK( cusolverMgDestroyMatrixDesc(descrA) );
    }

    if (gridA != NULL) {
        CUSOLVER_CHECK( cusolverMgDestroyGrid(gridA) );
    }

    if (0 > info) {
        char buffer[100];
        std::snprintf(buffer, sizeof(buffer), "%d-th parameter is wrong \n", -info);
        throw std::runtime_error(buffer);
    }

    CUDA_CHECK(cuda_status);
    CUSOLVER_CHECK(cuda_solver_status);

    CUDA_CHECK( hipMemGetInfo(&end_free_mem, &total_mem) );
    printf("end: %zu\n", end_free_mem);
    printf("difference: %zu\n", end_free_mem - start_free_mem);
    printf("difference: %zu\n", start_free_mem - end_free_mem);
}

static void signal_handler(int signum) {
    exit(signum);
}

void cusolverMgSyevd_export(
    torch::Tensor a, 
    torch::Tensor d,
    bool verbose
) {
    signal(SIGINT, signal_handler);

    if (a.dtype() != d.dtype())
        throw std::runtime_error("Both tensors must have same dtype");

    if (a.is_cuda() || d.is_cuda())
        throw std::runtime_error("Tensors must be on the Host, not device.");

    if (a.dim() != 2) 
        throw std::runtime_error("Dimension of tensor needs to be 2");

    if (a.size(0) != a.size(1)) 
        throw std::runtime_error("Matrix needs to be square");

    if (a.dtype() == torch::kFloat32) 
        return cusolverMgSyevd_template<float>(a, d, verbose);

    if (a.dtype() == torch::kFloat64) 
        return cusolverMgSyevd_template<double>(a, d, verbose);

    // If it gets here the dtype isn't supported
    throw std::runtime_error("Tensor needs to have dtype either float32 or float64");
}

void cusolverMgSyevd_workspace_query_export(
    int N,
    int num_devices,
    bool is_fp32,
    bool use_num_devices_visible,
    torch::Tensor workspace_num_elements,
    bool verbose
) {
    signal(SIGINT, signal_handler);
    
    if (workspace_num_elements.dtype() != torch::kInt64) {
        throw std::runtime_error("workspace_num_elements tensor needs to have dtype int64");
    }

    if (workspace_num_elements.numel() != 1) {
        throw std::runtime_error("workspace_num_elements tensor needs to have only one element");
    }

    int64_t num_workspace_elements;
    if (is_fp32) {
        cusolverMgSyevd_workspace_template<float>(N, num_devices, use_num_devices_visible, &num_workspace_elements, verbose);
    } else {
        cusolverMgSyevd_workspace_template<double>(N, num_devices, use_num_devices_visible, &num_workspace_elements, verbose);
    }
    *workspace_num_elements.data_ptr<int64_t>() = num_workspace_elements;
}